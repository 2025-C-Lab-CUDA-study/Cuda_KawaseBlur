#include "hip/hip_runtime.h"
﻿#include<hip/hip_runtime.h>
#include<>

#include <iostream>
#include"BmpUtile.h"


#define CUDA_CHECK(err) do { \
    if (err != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << " at line " << __LINE__ << std::endl; \
        exit(EXIT_FAILURE); \
    } \
} while (0)

#define CUDA_KERNEL_CHECK() CUDA_CHECK(hipGetLastError())


// Typedef ================

using uchar = unsigned char;


// Consts ================

constexpr int RATIO = 1;
constexpr int BLOCK = 16;
constexpr int BLURING_TIMES = 1;


__global__ void BilinearReduce(uchar* dstBuffer, size_t dstPitch, uchar* srcBuffer, size_t srcPitch, int dstWidth, int dstHeight, int srcWidth, int srcHeight)
{
    int src_pitch = srcPitch / sizeof(uchar);
    int dst_pitch = dstPitch / sizeof(uchar);

    for (
        int y = blockDim.y * blockIdx.y + threadIdx.y;
        y < dstHeight;
        y += blockDim.y * gridDim.y
        )
    {
        for (
            int x = blockDim.x * blockIdx.x + threadIdx.x;
            x < dstWidth;
            x += blockDim.x * gridDim.x
            )
        {
            int srcX = min(max(x * 2, 0), srcWidth - RATIO);
            int srcY = min(max(y * 2, 0), srcHeight - RATIO);

            uchar c1 = srcBuffer[srcX + src_pitch * srcY];
            uchar c2 = srcBuffer[srcX + RATIO + src_pitch * srcY];
            uchar c3 = srcBuffer[srcX + src_pitch * (srcY + RATIO)];
            uchar c4 = srcBuffer[srcX + RATIO + src_pitch * (srcY + RATIO)];

            dstBuffer[x + dst_pitch * y] = static_cast<uchar>((c1 + c2 + c3 + c4) * 0.25);
        }
    }
}

__global__ void BilinearIncrease(uchar* dstBuffer, size_t dstPitch, uchar* srcBuffer, size_t srcPitch, int dstWidth, int dstHeight, int srcWidth, int srcHeight)
{
    int src_pitch = srcPitch / sizeof(uchar);
    int dst_pitch = dstPitch / sizeof(uchar);

    for (
        int y = blockDim.y * blockIdx.y + threadIdx.y;
        y < dstHeight;
        y += blockDim.y * gridDim.y
        )
    {
        for (
            int x = blockDim.x * blockIdx.x + threadIdx.x;
            x < dstWidth;
            x += blockDim.x * gridDim.x
            )
        {
            int srcX = min(max(x / 2, 0), dstWidth - RATIO);
            int srcY = min(max(y / 2, 0), dstHeight - RATIO);

            uchar c1 = srcBuffer[srcX         + src_pitch * srcY];
            uchar c2 = srcBuffer[srcX + RATIO + src_pitch * srcY];
            uchar c3 = srcBuffer[srcX         + src_pitch * (srcY + RATIO)];
            uchar c4 = srcBuffer[srcX + RATIO + src_pitch * (srcY + RATIO)];

            dstBuffer[x + dst_pitch * y] = static_cast<uchar>((c1 + c2 + c3 + c4) * 0.25);
        }
    }
}


__global__ void KawaseBlur(uchar* dstBuffer, size_t dstPitch, hipTextureObject_t texObject, int offset, int width, int height)
{
    int dst_pitch = dstPitch / sizeof(uchar);

    for (int y = blockDim.y * blockIdx.y + threadIdx.y; y < height; y += blockDim.y * gridDim.y)
    {
        for (int x = blockDim.x * blockIdx.x + threadIdx.x; x < width; x += blockDim.x * gridDim.x)
        {
            int xmo = max(x - offset, 0);
            int ymo = max(y - offset, 0);
            int xpo = min(x + offset, width  - 1);
            int ypo = min(y + offset, height - 1);

            uchar c1 = tex2D<uchar>(texObject, xmo, ymo);
            uchar c2 = tex2D<uchar>(texObject, xpo, ymo);
            uchar c3 = tex2D<uchar>(texObject, xmo, ypo);
            uchar c4 = tex2D<uchar>(texObject, xpo, ypo);

            dstBuffer[x + dst_pitch * y] = static_cast<uchar>((c1 + c2 + c3 + c4) * 0.25);
        }
   }
}



int main(void)
{
    // Set Host data =========================================================================================

    uchar* h_rb = nullptr;
    uchar* h_gb = nullptr;
    uchar* h_bb = nullptr;
    int h_width, h_height;

    const char* path = "C:\\Users\\james\\Documents\\2025\\source_code\\lenna.bmp";
    if (!Bmp::BmpToRgbBuffers(path, &h_rb, &h_gb, &h_bb, h_width, h_height))
    {
        if (!h_rb) free(h_rb);
        if (!h_gb) free(h_gb);
        if (!h_bb) free(h_bb);
    }

    // Set Device data ========================================================================================

    int d_width = h_width;
    int d_height = h_height;
    size_t firstPitch, secondPitch;
    uchar* d_rb_first = nullptr, * d_rb_second = nullptr;
    uchar* d_gb_first = nullptr, * d_gb_second = nullptr;
    uchar* d_bb_first = nullptr, * d_bb_second = nullptr;

    CUDA_CHECK(hipMallocPitch(&d_rb_first, &firstPitch, sizeof(uchar) * d_width, d_height)); // fisrt buffer
    CUDA_CHECK(hipMallocPitch(&d_gb_first, &firstPitch, sizeof(uchar) * d_width, d_height));
    CUDA_CHECK(hipMallocPitch(&d_bb_first, &firstPitch, sizeof(uchar) * d_width, d_height));

    CUDA_CHECK(hipMallocPitch(&d_rb_second, &secondPitch, sizeof(uchar) * (d_width), d_height)); // second buffer
    CUDA_CHECK(hipMallocPitch(&d_gb_second, &secondPitch, sizeof(uchar) * (d_width), d_height));
    CUDA_CHECK(hipMallocPitch(&d_bb_second, &secondPitch, sizeof(uchar) * (d_width), d_height));

    CUDA_CHECK(hipMemcpy2D(d_rb_first, firstPitch, h_rb, sizeof(uchar) * h_width, sizeof(uchar) * h_width, h_height, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy2D(d_gb_first, firstPitch, h_gb, sizeof(uchar) * h_width, sizeof(uchar) * h_width, h_height, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy2D(d_bb_first, firstPitch, h_bb, sizeof(uchar) * h_width, sizeof(uchar) * h_width, h_height, hipMemcpyHostToDevice));

	// Set Texture data =========================================================================================
    
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uchar>();
    hipArray_t d_rb_array = nullptr;
    hipArray_t d_gb_array = nullptr;
    hipArray_t d_bb_array = nullptr;

    hipMallocArray(&d_rb_array, &channelDesc, d_width, d_height);
    hipMallocArray(&d_gb_array, &channelDesc, d_width, d_height);
    hipMallocArray(&d_bb_array, &channelDesc, d_width, d_height);

	hipResourceDesc rbResDesc;
    memset(&rbResDesc, 0, sizeof(rbResDesc));
	rbResDesc.resType = hipResourceTypeArray;
	rbResDesc.res.array.array = d_rb_array;

	hipResourceDesc gbResDesc;
    memset(&gbResDesc, 0, sizeof(gbResDesc));
	gbResDesc.resType = hipResourceTypeArray;
	gbResDesc.res.array.array = d_gb_array;

	hipResourceDesc bbResDesc;
    memset(&bbResDesc, 0, sizeof(bbResDesc));
	bbResDesc.resType = hipResourceTypeArray;
	bbResDesc.res.array.array = d_bb_array;

	hipTextureDesc rbTexDesc;
    memset(&rbTexDesc, 0, sizeof(rbTexDesc));
	rbTexDesc.addressMode[0] = hipAddressModeClamp; // x-axis
	rbTexDesc.addressMode[1] = hipAddressModeClamp; // y-axis
	rbTexDesc.filterMode = hipFilterModePoint; // Point filtering
	rbTexDesc.readMode = hipReadModeElementType; // Read as element type
	rbTexDesc.normalizedCoords = false; // Use absolute coordinates
   
    hipTextureDesc gbTexDesc;
    memset(&gbTexDesc, 0, sizeof(gbTexDesc));
    gbTexDesc.addressMode[0] = hipAddressModeClamp; // x-axis
    gbTexDesc.addressMode[1] = hipAddressModeClamp; // y-axis
    gbTexDesc.filterMode = hipFilterModePoint; // Point filtering
    gbTexDesc.readMode = hipReadModeElementType; // Read as element type
    gbTexDesc.normalizedCoords = false; // Use absolute coordinates

    hipTextureDesc bbTexDesc;
    memset(&bbTexDesc, 0, sizeof(bbTexDesc));
    bbTexDesc.addressMode[0] = hipAddressModeClamp; // x-axis
    bbTexDesc.addressMode[1] = hipAddressModeClamp; // y-axis
    bbTexDesc.filterMode = hipFilterModePoint; // Point filtering
    bbTexDesc.readMode = hipReadModeElementType; // Read as element type
    bbTexDesc.normalizedCoords = false; // Use absolute coordinates

	hipTextureObject_t texObjectRb, texObjectGb, texObjectBb;
    CUDA_CHECK(hipCreateTextureObject(&texObjectRb, &rbResDesc, &rbTexDesc, nullptr));
    CUDA_CHECK(hipCreateTextureObject(&texObjectGb, &gbResDesc, &gbTexDesc, nullptr));
    CUDA_CHECK(hipCreateTextureObject(&texObjectBb, &bbResDesc, &bbTexDesc, nullptr));

    // Run Bilinear and kawaseBlur ===============================================================================

    int offset = 1;
    int transWidth  = d_width  / 4;
    int transHeight = d_height / 4;
    
    dim3 block(BLOCK, BLOCK);

    for (int reduceKawase = 0; reduceKawase < BLURING_TIMES; ++reduceKawase)    // Run reduce image & kawase blur ---------------------------------------------
    {
        dim3 reduceGrid((transWidth + BLOCK - 1) / BLOCK, (transHeight + BLOCK - 1) / BLOCK);
        BilinearReduce << <reduceGrid, block >> > (d_rb_second, secondPitch, d_rb_first, firstPitch, d_width / 2, d_height / 2, d_width, d_height);
        CUDA_KERNEL_CHECK();
        BilinearReduce << <reduceGrid, block >> > (d_gb_second, secondPitch, d_gb_first, firstPitch, d_width / 2, d_height / 2, d_width, d_height);
        CUDA_KERNEL_CHECK();
        BilinearReduce << <reduceGrid, block >> > (d_bb_second, secondPitch, d_bb_first, firstPitch, d_width / 2, d_height / 2, d_width, d_height);
        CUDA_KERNEL_CHECK();

        hipDeviceSynchronize();
        hipDestroyTextureObject(texObjectRb);
        hipDestroyTextureObject(texObjectGb);
        hipDestroyTextureObject(texObjectBb);
        
        transWidth  /= 2;
        transHeight /= 2;
        d_width     /= 2;
        d_height    /= 2;

        CUDA_CHECK(hipMemcpy2DToArray(d_rb_array, 0, 0, d_rb_second, secondPitch, sizeof(uchar) * h_width, h_height, hipMemcpyDeviceToDevice));
        CUDA_CHECK(hipMemcpy2DToArray(d_gb_array, 0, 0, d_gb_second, secondPitch, sizeof(uchar) * h_width, h_height, hipMemcpyDeviceToDevice));
        CUDA_CHECK(hipMemcpy2DToArray(d_bb_array, 0, 0, d_bb_second, secondPitch, sizeof(uchar) * h_width, h_height, hipMemcpyDeviceToDevice));
            
        rbResDesc.res.array.array = d_rb_array;
        gbResDesc.res.array.array = d_gb_array;
        bbResDesc.res.array.array = d_bb_array;

        CUDA_CHECK(hipCreateTextureObject(&texObjectRb, &rbResDesc, &rbTexDesc, nullptr));
        CUDA_CHECK(hipCreateTextureObject(&texObjectGb, &gbResDesc, &gbTexDesc, nullptr));
        CUDA_CHECK(hipCreateTextureObject(&texObjectBb, &bbResDesc, &bbTexDesc, nullptr));

        dim3 kawaseGrid((transWidth + BLOCK - 1) / BLOCK, (transHeight + BLOCK - 1) / BLOCK);
        KawaseBlur << <kawaseGrid, block >> > (d_rb_first, firstPitch, texObjectRb, offset, d_width, d_height);
        CUDA_KERNEL_CHECK();
        KawaseBlur << <kawaseGrid, block >> > (d_gb_first, firstPitch, texObjectGb, offset, d_width, d_height);
        CUDA_KERNEL_CHECK();
        KawaseBlur << <kawaseGrid, block >> > (d_bb_first, firstPitch, texObjectBb, offset, d_width, d_height);
        CUDA_KERNEL_CHECK();

        ++offset;
    }


    // Check image status -----------------------------------------------------------------------------------------------------
    CUDA_CHECK(hipMemcpy2D(h_rb, sizeof(uchar) * h_width, d_rb_first, firstPitch, d_width, d_height, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy2D(h_gb, sizeof(uchar) * h_width, d_gb_first, firstPitch, d_width, d_height, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy2D(h_bb, sizeof(uchar) * h_width, d_bb_first, firstPitch, d_width, d_height, hipMemcpyDeviceToHost));
   
    const char* middlecheckPath = "C:\\Users\\james\\Documents\\2025\\source_code\\middleCheckLenna.bmp";
    if (!Bmp::RgbBuffersToBmp(middlecheckPath, h_rb, h_gb, h_bb, h_width, h_height))
    {
        std::cout << "Error : Writing bmp file failed";
    }

    --offset;
    for (int increaseKawase = 0; increaseKawase < BLURING_TIMES; ++increaseKawase)    // Run increasing image & kawase blur -------------------------------
    {
        dim3 grid((transWidth + BLOCK - 1) / BLOCK, (transHeight + BLOCK - 1) / BLOCK);

        BilinearIncrease <<<grid, block >>> (d_rb_second, secondPitch, d_rb_first, firstPitch, d_width * 2, d_height * 2, d_width, d_height);
        CUDA_KERNEL_CHECK();
        BilinearIncrease <<<grid, block >>> (d_gb_second, secondPitch, d_gb_first, firstPitch, d_width * 2, d_height * 2, d_width, d_height);
        CUDA_KERNEL_CHECK();
        BilinearIncrease <<<grid, block >>> (d_bb_second, secondPitch, d_bb_first, firstPitch, d_width * 2, d_height * 2, d_width, d_height);
        CUDA_KERNEL_CHECK();

        transWidth  *= 2;
        transHeight *= 2;
        d_width     *= 2;
        d_height    *= 2;

        CUDA_CHECK(hipMemcpy2DToArray(d_rb_array, 0, 0, d_rb_second, secondPitch, sizeof(uchar) * h_width, h_height, hipMemcpyDeviceToDevice));
        CUDA_CHECK(hipMemcpy2DToArray(d_gb_array, 0, 0, d_gb_second, secondPitch, sizeof(uchar) * h_width, h_height, hipMemcpyDeviceToDevice));
        CUDA_CHECK(hipMemcpy2DToArray(d_bb_array, 0, 0, d_bb_second, secondPitch, sizeof(uchar) * h_width, h_height, hipMemcpyDeviceToDevice));

        rbResDesc.res.array.array = d_rb_array;
        gbResDesc.res.array.array = d_gb_array;
        bbResDesc.res.array.array = d_bb_array;

        CUDA_CHECK(hipCreateTextureObject(&texObjectRb, &rbResDesc, &rbTexDesc, nullptr));
        CUDA_CHECK(hipCreateTextureObject(&texObjectGb, &gbResDesc, &gbTexDesc, nullptr));
        CUDA_CHECK(hipCreateTextureObject(&texObjectBb, &bbResDesc, &bbTexDesc, nullptr));

        dim3 kawaseGrid((transWidth + BLOCK - 1) / BLOCK, (transHeight + BLOCK - 1) / BLOCK);
        KawaseBlur <<<kawaseGrid, block >>> (d_rb_first, firstPitch, texObjectRb, offset, d_width, d_height);
        CUDA_KERNEL_CHECK();
        KawaseBlur <<<kawaseGrid, block >>> (d_gb_first, firstPitch, texObjectGb, offset, d_width, d_height);
        CUDA_KERNEL_CHECK();
        KawaseBlur <<<kawaseGrid, block >>> (d_bb_first, firstPitch, texObjectBb, offset, d_width, d_height);
        CUDA_KERNEL_CHECK();

        --offset;
    }


    // Store resized image ======================================================================================

    CUDA_CHECK(hipMemcpy2D(h_rb, sizeof(uchar) * h_width, d_rb_first, firstPitch, d_width, d_height, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy2D(h_gb, sizeof(uchar) * h_width, d_gb_first, firstPitch, d_width, d_height, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy2D(h_bb, sizeof(uchar) * h_width, d_bb_first, firstPitch, d_width, d_height, hipMemcpyDeviceToHost));

    const char* outPath2 = "C:\\Users\\james\\Documents\\2025\\source_code\\kawaseBluredLenna.bmp";
    if (!Bmp::RgbBuffersToBmp(outPath2, h_rb, h_gb, h_bb, h_width, h_height))
    {
        std::cout << "Error : Writing bmp file failed";
    }

    // free =====================================================================================================

    free(h_rb);
    free(h_gb);
    free(h_bb);
    hipFree(d_rb_first);
    hipFree(d_gb_first);
    hipFree(d_bb_first);
    hipFree(d_rb_second);
    hipFree(d_gb_second);
    hipFree(d_bb_second);
	hipDestroyTextureObject(texObjectRb);
	hipDestroyTextureObject(texObjectGb);
	hipDestroyTextureObject(texObjectBb);
    hipFreeArray(d_rb_array);
    hipFreeArray(d_gb_array);
    hipFreeArray(d_bb_array);

    return 0;
}